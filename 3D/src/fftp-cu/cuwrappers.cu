#include <hipfft/hipfft.h>

/*
  cuwrappers.cu

  CUDA wrappers. This includes those structures that allow Fortran
  to access C datatypes that might vary with different version of CUDA.
  There may be some overlap between these methods, and those in 
  cuda_utils.cu.

  2013 D. Rosenberg & P. Mininni
       ORNL NCCS
       email: mininni@df.uba.edu
*/


struct cudaDevicePropG /* must agree with cudaDevicePropG in cutypes.mod */
{
  int    canMapHostMemory;
  int    clockRate;
  int    computeMode;
  int    deviceOverlap;
  int    integrated;
  int    kernelExecTimeoutEnabled;
  int    major;
  int    maxGridSize[3];
  int    maxThreadsDim[3];
  int    maxThreadsPerBlock;
  int    memoryBusWidth;
  int    memoryClockRate;
  size_t memPitch;
  int    minor;
  int    multiProcessorCount;
  char   name[256];
  int    regsPerBlock;
  size_t sharedMemPerBlock;
  size_t textureAlignment;
  size_t totalConstMem;
  size_t totalGlobalMem;
  int    warpSize;
};

extern "C" {


/*
   void w_cudagetdeviceproperties_(cudaDevicePropG *devprop, int *idev)

   Retrieves standard set of device properties

   Since hipDeviceProp_t can add or delete types, we systematize them
   in the structure cudaDevicePropG, which is known both the Fortran
   and to the C:
*/
void w_cudagetdeviceproperties_(cudaDevicePropG *devprop, int *idev)
{
  hipDeviceProp_t prop;

  hipGetDeviceProperties(&prop,*idev);

  devprop->canMapHostMemory           = prop.canMapHostMemory;
  devprop->clockRate                  = prop.clockRate;
  devprop->computeMode                = prop.computeMode;
  devprop->deviceOverlap              = prop.deviceOverlap;
  devprop->integrated                 = prop.integrated;
  devprop->kernelExecTimeoutEnabled   = prop.kernelExecTimeoutEnabled;
  devprop->major                      = prop.major;
  memcpy(devprop->maxGridSize   ,prop.maxGridSize  , 3  *sizeof(int));
  memcpy(devprop->maxThreadsDim ,prop.maxThreadsDim, 3  *sizeof(int));
  devprop->maxThreadsPerBlock         = prop.maxThreadsPerBlock;
  devprop->memoryBusWidth             = prop.memoryBusWidth;
  devprop->memoryClockRate            = prop.memoryClockRate;
  devprop->memPitch                   = prop.memPitch;
  devprop->minor                      = prop.minor;
  devprop->multiProcessorCount        = prop.multiProcessorCount;
  memcpy(devprop->name          , prop.name        , 256*sizeof(char));
  devprop->regsPerBlock               = prop.regsPerBlock;
  devprop->sharedMemPerBlock          = prop.sharedMemPerBlock;
  devprop->textureAlignment           = prop.textureAlignment;
  devprop->totalConstMem              = prop.totalConstMem;
  devprop->totalGlobalMem             = prop.totalGlobalMem;
  devprop->warpSize                   = prop.warpSize;

} /* end of method w_cudagetdeviceproperties_ */


} /* extern C */

