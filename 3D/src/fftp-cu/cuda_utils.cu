#include "cuda_utils.hcu"
/***************************************************************
! cuda_utils.cu
!
! CUDA interface routines. Generally, contained here are those
! methods that require at least some CUDA. Those that are
! mainly interfaces to the CUDA RTL are contained in 'cuda_bindings.f90'
!
! 2011 Duane Rosenberg & Pablo D. Mininni
!      National Center for Atmospheric Research
!      e-mail: mininni@df.uba.ar 
!
***************************************************************/
#include <hipfft/hipfft.h>

extern "C" {

/* Memcpy methods: */
hipError_t cudaMemcpyHost2Dev( void *devdst, const void *hostsrc,  size_t count)
{
  hipError_t iret;
  iret = hipMemcpy( devdst, hostsrc, count,  hipMemcpyHostToDevice ) ;
  return iret;
}

hipError_t cudaMemcpyDev2Host( void *hostdst, const void *devsrc,  size_t count)
{
  hipError_t iret;
  iret = hipMemcpy( hostdst , devsrc, count,  hipMemcpyDeviceToHost ); 
  return iret;
}

/* New interface for hipfftPlanMany: */
int  MycufftPlanMany(hipfftHandle *plan, int rank, int *n, int *inembed,
                       int istride, int idist, int *onembed, int ostride,
                       int odist, hipfftType type, int batch) 
{
  int iret;
  
  if ( inembed != NULL && inembed[0] <= 0 ) {
    iret = (int)hipfftPlanMany(plan,rank,n,NULL,istride,idist,
                                     NULL,ostride,odist,type,batch);
  } else {
    iret = (int)hipfftPlanMany(plan,rank,n,inembed,istride,idist,
                                           onembed,ostride,odist,type,batch);
  }
  return iret;
}


} /* end, extern "C" interface */


