#include "cuda_utils.hcu"

extern "C" {

/* Memcpy methods: */
hipError_t cudaMemcpyHost2Dev( void *devdst, const void *hostsrc,  size_t count)
{
  hipError_t iret;
  iret = hipMemcpy( devdst, hostsrc, count,  hipMemcpyHostToDevice ) ;
  return iret;
}

hipError_t cudaMemcpyDev2Host( void *hostdst, const void *devsrc,  size_t count)
{
  return hipMemcpy( hostdst , devsrc, count,  hipMemcpyDeviceToHost ); 
}


} /* end, extern "C" interface */


