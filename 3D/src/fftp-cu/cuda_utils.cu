#include "cuda_utils.hcu"
/***************************************************************
! cuda_utils.cu
!
! CUDA interface routines. Generally, contained here are those
! methods that require at least some CUDA. Those that are
! mainly interfaces to the CUDA RTL are contained in 'cuda_bindings.f90'
!
! 2011 Duane Rosenberg & Pablo D. Mininni
!      National Center for Atmospheric Research
!      e-mail: mininni@df.uba.ar 
!
***************************************************************/
#include <hipfft/hipfft.h>

extern "C" {

/* Memcpy methods: */
hipError_t cudaMemcpyHost2Dev( void *devdst, const void *hostsrc,  size_t count)
{
  hipError_t iret;
  iret = hipMemcpy( devdst, hostsrc, count,  hipMemcpyHostToDevice ) ;
  return iret;
}

hipError_t cudaMemcpyDev2Host( void *hostdst, const void *devsrc,  size_t count)
{
  return hipMemcpy( hostdst , devsrc, count,  hipMemcpyDeviceToHost ); 
}

} /* end, extern "C" interface */


